// from https://yuki67.github.io/post/cuda_time/

#include <hip/hip_runtime.h>
#include <stdio.h>

#include "cudaTimer.h"
#include <vector>
#include <iostream>
#include <fstream>
#include <numeric>
#include <algorithm>
#include <random>

#include <assert.h>

#define all(a) (a).begin(),(a).end()

// グローバル変数
// コピー用変数の準備
float *a_h, *b_h;	// ホストデータ
float *a_d, *b_d;	// デバイスデータ
long long n, nBytes; // n:データ数 / nBytes:総データ量(bytes)
// vector<long long> n_list = {160LL, 1024, 3000, 160LL*180*360, 1024LL*1024*1200, /*3000LL*3500*360*/};
// vector<int> n_list = {1024 / sizeof(float), 512 * 1024 / sizeof(float), 1024 * 1024 / sizeof(float), 128 * 1024 * 1024 / sizeof(float)};

// 結果格納変数
vector<float> h2d, d2d, d2h;
// その他制御変数
bool flg = false; // printf起動制御
int itr_times = 1000; // 一つの条件に対する実験の繰り返し回数
ofstream ofs;
random_device seed_gen;
uniform_real_distribution<float> dist(-1.0,1.0);

void setup(){
	mt19937 engine(seed_gen());

	a_h = (float *)malloc(nBytes); // これで配列として確保できる
	b_h = (float *)malloc(nBytes);
	hipMalloc((void **) &a_d, nBytes);
	hipMalloc((void **) &b_d, nBytes);

	fill(a_h, a_h + n, 1.3);

	/*
	for(long long i = 0; i < n; i++){
		a_h[i] = dist(engine);
	}
	*/

	return;
}

void measure(){
	// 計測
	CudaTimer timer;
	// 1. H2D
	timer.begin();
	// ~~~計測対象の動作を記述~~~
	hipMemcpy(a_d, a_h, nBytes, hipMemcpyHostToDevice);
	h2d.emplace_back( timer.stop_and_report("H2D",flg) );

	// 2. D2D
	timer.begin();
	// ~~~計測対象の動作を記述~~~
	hipMemcpy(b_d, a_d, nBytes, hipMemcpyDeviceToDevice);
	d2d.emplace_back( timer.stop_and_report("D2D",flg) );

	// 3.D2H
	timer.begin();
	// ~~~計測対象の動作を記述~~~
	hipMemcpy(b_h, b_d, nBytes, hipMemcpyDeviceToHost);
	d2h.emplace_back( timer.stop_and_report("D2H",flg) );

	// 正誤チェック
	for(long long i = 0; i < n; i++){
		assert(a_h[i] == b_h[i]);
	}

	return;
}

void memFree(){
	// データ解放
	free(a_h);
	free(b_h);
	hipFree(a_d);
	hipFree(b_d);

	return;
}

void csv_out() {ofs << endl;}

template <typename Head, typename... Tail>
void csv_out(Head H, Tail... T){
	ofs << H;
	csv_out(T...);
}
#define csvo(...) csv_out(__VA_ARGS__)

void put_csv(long long Bytes){
	// エラーチェック
	assert(h2d.size() == d2d.size() && d2d.size() == d2h.size());
	
	/*
	// データ入力
	for(long long ind = 0; ind < h2d.size(); ++ind){
		ofs << h2d.at(ind) << "," << "," << d2d.at(ind) << "," << "," << d2h.at(ind) << "," << endl;
	}
	*/
	
	// 平均データ入力
	float h2d_ave = accumulate(h2d.begin(),h2d.end(),0.0f) / h2d.size();
	float d2d_ave = accumulate(d2d.begin(),d2d.end(),0.0f) / d2d.size();
	float d2h_ave = accumulate(d2h.begin(),d2h.end(),0.0f) / d2h.size();
	//ofs << "," << "Ave." <<endl;
	//ofs << h2d_ave << "," << "," << d2d_ave << "," << "," << d2h_ave << "," << endl;
	
	// 中央値データ入力
	sort(all(h2d));	sort(all(d2d));	sort(all(d2h));
	size_t med_ind = h2d.size() / 2;
	float h2d_med = (h2d.size() % 2 == 0
    ? static_cast<float>(h2d[med_ind] + h2d[med_ind - 1]) / 2
	: h2d[med_ind]);
	float d2d_med = (d2d.size() % 2 == 0
    ? static_cast<float>(d2d[med_ind] + d2d[med_ind - 1]) / 2
	: d2d[med_ind]);
	float d2h_med = (d2h.size() % 2 == 0
    ? static_cast<float>(d2h[med_ind] + d2h[med_ind - 1]) / 2
	: d2h[med_ind]);
	//ofs << "," << "Med." <<endl;
	//ofs << h2d_med << "," << "," << d2d_med << "," << "," << d2h_med << "," << endl;

	// データ書き込み
	csvo(Bytes/1024,",,",h2d_ave,",",d2d_ave,",",d2h_ave,",,",h2d_med,",",d2d_med,",",d2h_med);
}

int main() {

	string data_name = "time_plot_data.csv";
	ofs.open(data_name);
	// 項目の入力
	csvo("(KBytes)\\(msec.),","<Ave.>,","H2D,","D2D,","D2H,","<Med.>,","H2D,","D2D,","D2H");
	//ofs << "H2D" << "," << "," << "D2D" << "," << "," << "D2H" << "," << endl;

	// for(long long ele : n_list){
	long long ele = 256LL;
	long long add = 256LL;
	long long base = 10LL;
	long long div = 1024LL;
	bool flg = false;
	while(ele < 1024LL*1024*512){	//1GBまで?
		n = ele;
		nBytes = n * sizeof(float);
		if(nBytes / (1024 * 1024) > 0){
			cout << "transport data size : " << nBytes / (1024 * 1024) << "[M Bytes]" << endl;
			// data_name = to_string(nBytes / (1024 * 1024)) + "M_Bytes_measure.csv";
		}
		else if(nBytes / 1024 > 0){
			cout << "transport data size : " << nBytes / 1024 << "[K Bytes]" << endl;
			// data_name = to_string(nBytes / 1024) + "K_Bytes_measure.csv";
		}
		else{
			cout << "transport data size : " << nBytes << "[Bytes]" << endl;
			// data_name = to_string(nBytes) + "Bytes_measure.csv";
		}

		h2d.clear();
		d2d.clear();
		d2h.clear();

		for(int i = 0; i < itr_times; ++i){
			setup();
			measure();
			memFree();
		}
	
		cout << "finished" << "\n";
		put_csv(nBytes);

		if(nBytes / div >= base){
			base *= 10;
			add *= 10;
		}
		ele += add;
		if(!flg && ele * sizeof(float) / 1024 > 1000){
			flg = true;
			div *= 1024;
			base = 10;
			add = 256LL * 1024;
			ele = 256LL * 1024;
		}
	}

	ofs.close();

	return 0;
	/*
	nBytes = n * sizeof(float);
	a_h = (float *)malloc(nBytes); // これで配列として確保できる
	b_h = (float *)malloc(nBytes);
	hipMalloc((void **) &a_d, nBytes);
	hipMalloc((void **) &b_d, nBytes);

	for(int i = 0; i < n; i++){
		a_h[i] = 100.0f + i;
	}
	*/

	/*
	// 計測
	CudaTimer timer;
	// 1. H2D
	timer.begin();
	// ~~~計測対象の動作を記述~~~
	hipMemcpy(a_d, a_h, nBytes, hipMemcpyHostToDevice);
	timer.stop_and_report("H2D");

	// 2. D2D
	timer.begin();
	// ~~~計測対象の動作を記述~~~
	hipMemcpy(b_d, a_d, nBytes, hipMemcpyDeviceToDevice);
	timer.stop_and_report("D2D");

	// 3.D2H
	timer.begin();
	// ~~~計測対象の動作を記述~~~
	hipMemcpy(b_h, b_d, nBytes, hipMemcpyDeviceToHost);
	timer.stop_and_report("D2H");

	// 正誤チェック
	for(int i = 0; i < n; i++){
		assert(a_h[i] == b_h[i]);
	}
	*/

	/*
	// データ解放
	free(a_h);
	free(b_h);
	hipFree(a_d);
	hipFree(b_d);
	*/
}
